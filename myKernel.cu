#include "hip/hip_runtime.h"
#include "myKernel.cuh"
#include <hip/hip_runtime.h>
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA ERROR::" << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << " \n";
        hipDeviceReset();
        exit(99);
    }
}
__global__
void add(int H, int W, ull* d_grid, ull* d_rules)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
}

__global__ 
void propogateCuda_multiarray(int H, int W, int M, int center_row, int center_col, ull* d_grid
, ull* d_grid_left, ull* d_grid_right, ull* d_grid_up, ull* d_grid_down)
{
    
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    //int stride = blockDim.x;
    int stride = 10000;
    //__shared__ unsigned long long d_rules_total[64*4];
    __shared__ int canexit;
    //__shared__ unsigned long long bottom_top_rules[64];
    //__shared__ unsigned long long left_right_rules[64];
    //__shared__ unsigned long long right_left_rules[64];
    //__shared__ double weights[64];
    //__shared__ double weightLogweights[64];
    //for(int i=0;i<4*M;i++)
    //{
        //d_rules_total[i] = d_rules[i];
        //top_bottom_rules[i] = wfc_solver->top_bottom_rules[i];
        //bottom_top_rules[i] = wfc_solver->bottom_top_rules[i];
        //left_right_rules[i] = wfc_solver->left_right_rules[i];
        //right_left_rules[i] = wfc_solver->right_left_rules[i];
        //weights[i] = wfc_solver->weights[i];
        //weightLogweights[i] = wfc_solver->weightLogweights[i];
    //}
    canexit = 0;
    int col = index % W;
    int row = index / W;
    int left = col - 1;
    int right = col + 1;
    int up = row - 1;
    int down = row + 1;
    bool leftcheck = left >= 0;
    bool rightcheck = right < W;
    bool upcheck = up >= 0 ;
    bool downcheck = down < H;
    ull my = d_grid[index];
    ull result = my;
    ull sp_left = 1<<M - 1, sp_right = 1<<M - 1, sp_up = 1<<M - 1, sp_down = 1<<M - 1;
    //__syncthreads();
    


    int c = H+W;

    
    while((c--))
    {
        for(int i=index;i<H*W;i+=stride)
        {
            col = i % W; row = i / W;
            left = col - 1; right = col + 1; up = row - 1; down = row + 1;
            leftcheck = left >= 0; rightcheck = right < W; upcheck = up >= 0; downcheck = down < H;
            my = d_grid[i];
            result = my;
            sp_left = leftcheck ? d_grid_left[row* W + left] : 1<<M - 1;
            sp_right = rightcheck ? d_grid_right[row* W + right] : 1<<M - 1;
            sp_up = upcheck ? d_grid_up[up*W + col] : 1<<M - 1;
            sp_down = downcheck ? d_grid_down[down*W + col] : 1<<M - 1;

            ull vaild_state = 0;
            for (ull j = 0; j < M ; j++)
            {
                vaild_state = ((sp_left >> j) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j]) : vaild_state;
            }
            result = leftcheck ? result & vaild_state : result;
            vaild_state = 0;
            for (ull j = 0; j < M ; j++)
            {
                vaild_state = ((sp_right >> j) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j]) : vaild_state;
            }
            result = rightcheck ? result & vaild_state : result;
            vaild_state = 0;
            for (ull j = 0; j < M ; j++)
            {   
                vaild_state = ((sp_up >> j) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j]) : vaild_state;
            }
            result = upcheck ? result & vaild_state : result;
            vaild_state = 0;
            for (ull j = 0; j < M ; j++)
            {
                vaild_state = ((sp_down >> j) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j]) : vaild_state;
            }
            result = downcheck ? result & vaild_state : result;
            d_grid[i] = result;
            d_grid_left[i] = result;
            d_grid_right[i] = result;
            d_grid_down[i] = result;
            d_grid_up[i] = result;
            if(my > result) canexit = 1;
            //canexit = my == result ? canexit : 1;
        }
        __syncthreads();
        if(canexit == 0)break;
        canexit = 0;
    }    
}

__global__ 
void propogateCuda(int H, int W, int M, int center_row, int center_col, ull* d_grid)
{
    int _col = blockIdx.x * blockDim.x + threadIdx.x;
    int _row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = _row  * blockDim.x + threadIdx.x;
    int stride = 1024;
    
    __shared__ int canexit;
    canexit = 0;
    int col = index % W;
    int row = index / W;
    int left = col - 1;
    int right = col + 1;
    int up = row - 1;
    int down = row + 1;
    bool leftcheck = left >= 0;
    bool rightcheck = right < W;
    bool upcheck = up >= 0 ;
    bool downcheck = down < H;
    ull my = d_grid[index];
    ull result = my;
    ull sp_left = 1<<M - 1, sp_right = 1<<M - 1, sp_up = 1<<M - 1, sp_down = 1<<M - 1;

    


    int c = H+W;

    
    while((c--))
    {
        for(int i=index;i<H*W;i+=stride)
        {
            col = i % W; row = i / W;
            left = col - 1; right = col + 1; up = row - 1; down = row + 1;
            leftcheck = left >= 0; rightcheck = right < W; upcheck = up >= 0; downcheck = down < H;
            my = d_grid[i];
            result = my;

            sp_left = leftcheck ? d_grid[row* W + left] : 1<<M - 1;
            ull vaild_state = 0;
            for (int j = 0; j < M ; j+=8)
            {
                vaild_state = ((sp_left >> j) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j]) : vaild_state;
                vaild_state = ((sp_left >> j+1) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j+1]) : vaild_state;
                vaild_state = ((sp_left >> j+2) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j+2]) : vaild_state;
                vaild_state = ((sp_left >> j+3) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j+3]) : vaild_state;
                vaild_state = ((sp_left >> j+4) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j+4]) : vaild_state;
                vaild_state = ((sp_left >> j+5) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j+5]) : vaild_state;
                vaild_state = ((sp_left >> j+6) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j+6]) : vaild_state;
                vaild_state = ((sp_left >> j+7) & 1ull) && leftcheck ? (vaild_state | d_rules[M * 2 + j+7]) : vaild_state;
            }
            result = leftcheck ? result & vaild_state : result;
            
            sp_right = rightcheck ? d_grid[row* W + right] : 1<<M - 1;
            vaild_state = 0;
            for (int j = 0; j < M ; j+=8)
            {
                vaild_state = ((sp_right >> j) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j]) : vaild_state;
                vaild_state = ((sp_right >> j+1) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j+1]) : vaild_state;
                vaild_state = ((sp_right >> j+2) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j+2]) : vaild_state;
                vaild_state = ((sp_right >> j+3) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j+3]) : vaild_state;
                vaild_state = ((sp_right >> j+4) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j+4]) : vaild_state;
                vaild_state = ((sp_right >> j+5) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j+5]) : vaild_state;
                vaild_state = ((sp_right >> j+6) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j+6]) : vaild_state;
                vaild_state = ((sp_right >> j+7) & 1ull) && rightcheck ? (vaild_state | d_rules[M * 3 + j+7]) : vaild_state;
            }
            result = rightcheck ? result & vaild_state : result;
            
            
            sp_up = upcheck ? d_grid[up*W + col] : 1<<M - 1;
            vaild_state = 0;
            for (int j = 0; j < M ; j+=8)
            {   
                vaild_state = ((sp_up >> j) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j]) : vaild_state;
                vaild_state = ((sp_up >> j+1) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j+1]) : vaild_state;
                vaild_state = ((sp_up >> j+2) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j+2]) : vaild_state;
                vaild_state = ((sp_up >> j+3) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j+3]) : vaild_state;
                vaild_state = ((sp_up >> j+4) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j+4]) : vaild_state;
                vaild_state = ((sp_up >> j+5) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j+5]) : vaild_state;
                vaild_state = ((sp_up >> j+6) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j+6]) : vaild_state;
                vaild_state = ((sp_up >> j+7) & 1ull) && upcheck ? (vaild_state | d_rules[M * 0 + j+7]) : vaild_state;
            }
            result = upcheck ? result & vaild_state : result;
            
            sp_down = downcheck ? d_grid[down*W + col] : 1<<M - 1;
            vaild_state = 0;
            for (int j = 0; j < M ; j+=8)
            {
                vaild_state = ((sp_down >> j) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j]) : vaild_state;
                vaild_state = ((sp_down >> j+1) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j+1]) : vaild_state;
                vaild_state = ((sp_down >> j+2) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j+2]) : vaild_state;
                vaild_state = ((sp_down >> j+3) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j+3]) : vaild_state;
                vaild_state = ((sp_down >> j+4) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j+4]) : vaild_state;
                vaild_state = ((sp_down >> j+5) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j+5]) : vaild_state;
                vaild_state = ((sp_down >> j+6) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j+6]) : vaild_state;
                vaild_state = ((sp_down >> j+7) & 1ull) && downcheck ? (vaild_state | d_rules[M * 1 + j+7]) : vaild_state;
            }
            result = downcheck ? result & vaild_state : result;
            d_grid[i] = result;
            if(my > result) canexit = 1;
            //canexit = my == result ? canexit : 1;
        }
        __syncthreads();
        if(canexit == 0)break;
        canexit = 0;
    }    
}

__global__ 
void propogateCuda_col(int H, int W, int M, int center_row, int center_col, ull* d_grid, ull* d_grid_col)
{
    
    int index = threadIdx.x;
    int stride = blockDim.x;
    //__shared__ unsigned long long d_rules_total[64*4];
    __shared__ int canexit;
    //__shared__ unsigned long long bottom_top_rules[64];
    //__shared__ unsigned long long left_right_rules[64];
    //__shared__ unsigned long long right_left_rules[64];
    //__shared__ double weights[64];
    //__shared__ double weightLogweights[64];
    //for(int i=0;i<4*M;i++)
    //{
        //d_rules_total[i] = d_rules[i];
        //top_bottom_rules[i] = wfc_solver->top_bottom_rules[i];
        //bottom_top_rules[i] = wfc_solver->bottom_top_rules[i];
        //left_right_rules[i] = wfc_solver->left_right_rules[i];
        //right_left_rules[i] = wfc_solver->right_left_rules[i];
        //weights[i] = wfc_solver->weights[i];
        //weightLogweights[i] = wfc_solver->weightLogweights[i];
    //}
    canexit = 0;
    int col = index % W;
    int row = index / W;
    int left = col - 1;
    int right = col + 1;
    int up = row - 1;
    int down = row + 1;
    bool leftcheck = left >= 0;
    bool rightcheck = right < W;
    bool upcheck = up >= 0 ;
    bool downcheck = down < H;
    ull my = d_grid[index];
    ull result = my;
    ull sp_left = 1<<M - 1, sp_right = 1<<M - 1, sp_up = 1<<M - 1, sp_down = 1<<M - 1;
    //__syncthreads();
    


    int c = H+W;

    
    while((c--))
    {
        for(int i=index;i<H*W;i+=stride)
        {
            col = i % W; row = i / W;
            left = col - 1; right = col + 1; up = row - 1; down = row + 1;
            leftcheck = left >= 0; rightcheck = right < W; upcheck = up >= 0; downcheck = down < H;
            my = d_grid[i];
            result = my;
            sp_left = leftcheck ? d_grid[row* W + left] : 1<<M - 1;
            sp_right = rightcheck ? d_grid[row* W + right] : 1<<M - 1;
            sp_up = upcheck ? d_grid_col[up + col * H] : 1<<M - 1;
            sp_down = downcheck ? d_grid_col[down + col*H] : 1<<M - 1;

            ull vaild_state = 0;
            for (ull j = 0; j < M && leftcheck; j++)
            {
                vaild_state = ((sp_left >> j) & 1ull) ? (vaild_state | d_rules[M * 2 + j]) : vaild_state;
            }
            result = leftcheck ? result & vaild_state : result;
            vaild_state = 0;
            for (ull j = 0; j < M && rightcheck; j++)
            {
                vaild_state = ((sp_right >> j) & 1ull) ? (vaild_state | d_rules[M * 3 + j]) : vaild_state;
            }
            result = rightcheck ? result & vaild_state : result;
            vaild_state = 0;
            for (ull j = 0; j < M && upcheck; j++)
            {   
                vaild_state = ((sp_up >> j) & 1ull) ? (vaild_state | d_rules[M * 0 + j]) : vaild_state;
            }
            result = upcheck ? result & vaild_state : result;
            vaild_state = 0;
            for (ull j = 0; j < M && downcheck; j++)
            {
                vaild_state = ((sp_down >> j) & 1ull) ? (vaild_state | d_rules[M * 1 + j]) : vaild_state;
            }
            result = downcheck ? result & vaild_state : result;
            d_grid[i] = result;
            d_grid_col[col*H+row] = result;
            if(my > result) canexit = 1;
            //canexit = my == result ? canexit : 1;
        }
        __syncthreads();
        if(canexit == 0)break;
        canexit = 0;
    }    
}

__global__ 
void propogateCuda_multiblock(int H, int W, int M, int center_row, int center_col, ull* d_grid, int *block_src, int *canexit)
{
    (*block_src) = 0;
    (*canexit) = 0;

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    //int stride = blockDim.x;
    //__shared__ int canexit;
    int col = index % W;
    int row = index / W;
    int left = col - 1;
    int right = col + 1;
    int up = row - 1;
    int down = row + 1;
    bool leftcheck = left >= 0;
    bool rightcheck = right < W;
    bool upcheck = up >= 0 ;
    bool downcheck = down < H;
    ull my = d_grid[index];
    ull result = my;
    ull sp_left = 1<<M - 1, sp_right = 1<<M - 1, sp_up = 1<<M - 1, sp_down = 1<<M - 1;
    int c = H+W;
    while((c--))
    {
        col = index % W; row = index / W;
        left = col - 1; right = col + 1; up = row - 1; down = row + 1;
        leftcheck = left >= 0; rightcheck = right < W; upcheck = up >= 0; downcheck = down < H;
        my = d_grid[index];
        result = my;
        sp_left = leftcheck ? d_grid[row* W + left] : 1<<M - 1;
        sp_right = rightcheck ? d_grid[row* W + right] : 1<<M - 1;
        sp_up = upcheck ? d_grid[up*W + col] : 1<<M - 1;
        sp_down = downcheck ? d_grid[down*W + col] : 1<<M - 1;

        ull vaild_state = 0;
        for (ull j = 0; j < M && leftcheck; j++)
        {
            vaild_state = ((sp_left >> j) & 1ull) ? (vaild_state | d_rules[M * 2 + j]) : vaild_state;
        }
        result = leftcheck ? result & vaild_state : result;
        vaild_state = 0;
        for (ull j = 0; j < M && rightcheck; j++)
        {
            vaild_state = ((sp_right >> j) & 1ull) ? (vaild_state | d_rules[M * 3 + j]) : vaild_state;
        }
        result = rightcheck ? result & vaild_state : result;
        vaild_state = 0;
        for (ull j = 0; j < M && upcheck; j++)
        {   
            vaild_state = ((sp_up >> j) & 1ull) ? (vaild_state | d_rules[M * 0 + j]) : vaild_state;
        }
        result = upcheck ? result & vaild_state : result;
        vaild_state = 0;
        for (ull j = 0; j < M && downcheck; j++)
        {
            vaild_state = ((sp_down >> j) & 1ull) ? (vaild_state | d_rules[M * 1 + j]) : vaild_state;
        }
        result = downcheck ? result & vaild_state : result;
        d_grid[index] = result;
        if(my > result) (*canexit) = 1;
        //canexit = my == result ? canexit : 1;
    }
    //if(threadIdx.x == 0)
    //    atomicAdd(block_src,1);

    __syncthreads();
    //while((*block_src) != H)continue;

    if((*canexit) == 0)return;
    (*canexit) = 0;    
}

CudaWFC::CudaWFC(int H, int W,  shared_ptr<Rule> rules, int selection):WFC(H, W, rules,selection){
    assert(rules->M <= 64);

    auto init = sp_to_bits(rules->initValue());
    h_grid = (ull*)calloc(H * W, sizeof(ull));
    for(int i=0; i< H*W; i++)
        h_grid[i] = init;
    h_rules = (ull*)calloc(rules->M * 4, sizeof(ull));

    M = rules->M;
    for (int i = 0; i < rules->M; i++)
    {
        h_rules[M * 0 + i] = sp_to_bits(rules->top_bottom_rules[i]);
        h_rules[M * 1 + i] = sp_to_bits(rules->bottom_top_rules[i]);
        h_rules[M * 2 + i] = sp_to_bits(rules->left_right_rules[i]);
        h_rules[M * 3 + i] = sp_to_bits(rules->right_left_rules[i]);
    }
    hipMalloc((void**)&d_grid, sizeof(ull) * H*W);
    hipMemcpy(d_grid, h_grid, sizeof(ull) * H*W, hipMemcpyHostToDevice);
    //hipMalloc((void**)&d_grid_backup, sizeof(ull) * H*W);
    //hipMalloc((void**)&d_rules, sizeof(ull) * rules->M * 4);
    //hipMemcpy(d_grid, h_grid, sizeof(ull) * H*W, hipMemcpyHostToDevice);
    //hipMemcpy(d_rules, h_rules, sizeof(ull) * rules->M * 4, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_rules), h_rules, sizeof(ull) * rules->M * 4);
    //hipMemcpy(d_grid_backup, h_grid, sizeof(ull) * H*W, hipMemcpyHostToDevice);
}


RETURN_STATE CudaWFC::collapse(Position &position, RandomGen &random, bool print_step)
{
    int row = position.first, col = position.second;
    int index = row * W + col;
    // Copy a single value back to the host
    ull state = h_grid[index];
    if(state == 0){ // There is no pattern available for a cell
        return FAILED;
    }

    int collapsed_state = -1;
    auto size = std::popcount(state); // count how many 1 in binary representation
    if(selection <= 2){
        // keep n-th 1, the other 1 to 0 
        auto n = findNthSetBit(state, 1 + (random.randomInt() % size));
        // collapse to one pattern
        state = 1ull << n;
        collapsed_state = n;
    }
    else{
        throw std::logic_error("Method not yet implemented");
    }

    // Copy a single value back to the device
    h_grid[index] = state;
    if(print_step){
        std::cout << position.first << " " << position.second;
        std::cout << " collapse to " << collapsed_state << "\n";
        printGrid();
        std::cout << "\n";
    }

    return OK;
};

template <typename Set>
Position CudaWFC::impl_selectOneCell(Set &unobserved, RandomGen &random){
    if(selection <= 1){  // first element of order_set, unorderd_set
        auto position_it = unobserved.begin();
        return *position_it;
    }
    else if (selection == 2){ // full random
        auto position_it = unobserved.begin();
        std::advance(position_it, random.randomInt() % unobserved.size());
        return *position_it;
    }
    else{
        // implement other methods e.g. min entropy selection
        // or cuda version
        throw std::logic_error("Method not yet implemented");
    }
}
struct comapre
{
    __device__
    bool operator()(ull x) const {
        return __popcll(x)!=1;
    }
};
template <typename Set>
void CudaWFC::impl_propogate(Set &unobserved, Position &position, bool print_process){
    // TODO
    hipMemcpy(d_grid, h_grid, sizeof(ull) * H*W, hipMemcpyHostToDevice);
    //hipMemcpy(d_grid, h_grid, sizeof(ull) * H*W, hipMemcpyHostToDevice);
    int row = position.first, col = position.second;
    int index = row * W + col;
    int block = H*W/1024;
    if(block<=0)block = 1;
    propogateCuda<<<block,1024>>>(H,W, M, row, col,d_grid);
    //add<<<1,1>>>(H,W,d_grid,d_rules);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Kernel Error: " << hipGetErrorString(error) << std::endl;
        return ;
    }
    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(h_grid, d_grid, sizeof(ull) * H*W, hipMemcpyDeviceToHost);
    for (auto it = unobserved.begin(); it != unobserved.end(); ) {
        Position pos = (*it);
        row = pos.first, col = pos.second;
        index = row * W + col;
        if (std::popcount(h_grid[index])==1) {
            it = unobserved.erase(it);
        }
        else {
            ++it;
        }
    }
    
}